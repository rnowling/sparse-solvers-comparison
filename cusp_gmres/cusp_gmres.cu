#include "hip/hip_runtime.h"
#include <cusp/csr_matrix.h>
#include <cusp/monitor.h>
#include <cusp/krylov/gmres.h>
#include <cusp/io/matrix_market.h>

#include <fstream>
#include <iostream>
#include <sys/time.h>

void load_vector(const char* flname, cusp::array1d<float, cusp::host_memory> &v)
{
  std::ifstream input_data;
  input_data.open(flname);

  int n_elements;
  input_data >> n_elements;

  v.resize(n_elements);

  for(int i = 0; i < n_elements; i++)
    {
      float entry;
      input_data >> entry;
      v[i] = entry;
    }

  input_data.close();
}

void write_vector(const char* flname, cusp::array1d<float, cusp::host_memory> &v)
{
  std::ofstream output_data;
  output_data.open(flname, std::ofstream::out | std::ofstream::trunc);

  output_data << v.size() << std::endl;
  for(int i = 0; i < v.size(); i++)
    {
      output_data << v[i] << std::endl;
    }
  
  output_data.close();  
}

long int elapsed_time_ms(struct timespec &start, struct timespec &end)
{
  return (end.tv_sec * 1000 + end.tv_nsec / (1000 * 1000)) -
    (start.tv_sec * 1000 + start.tv_nsec / (1000 * 1000));
}


int main(int argc, char** argv)
{
  if(argc != 4)
    {
      std::cout << "Usage: " << argv[0] << " <matrix_flname> <input_vector_flname> <output_vector_flname>" << std::endl;
      return 1;
    }
  
  // create an empty sparse matrix structure (CSR format)
  cusp::csr_matrix<int, float, cusp::host_memory> A_host;
  // read matrix
  cusp::io::read_matrix_market_file(A_host, argv[1]);

  // create empty array
  cusp::array1d<float, cusp::host_memory> b_host(A_host.num_cols, 0);
  // read vector
  load_vector(argv[2], b_host);

  std::cout << "Matrix dimensions: " << A_host.num_rows << " " << A_host.num_cols << std::endl;
  std::cout << "Vector length : " << b_host.size() << std::endl;      

  struct timespec copy_start;
  struct timespec copy_end;
  struct timespec exec_start;
  struct timespec exec_end;
  
  clock_gettime(CLOCK_MONOTONIC, &copy_start);
  cusp::csr_matrix<int, float, cusp::device_memory> A(A_host);
  cusp::array1d<float, cusp::device_memory> b(b_host);  
  cusp::array1d<float, cusp::device_memory> x(A.num_rows, 0);
  // set preconditioner (identity)
  cusp::identity_operator<float, cusp::device_memory> M(A_host.num_rows, A_host.num_rows);
  clock_gettime(CLOCK_MONOTONIC, &copy_end);
  
  // set stopping criteria:
  //  iteration_limit    = 5000
  //  relative_tolerance = 1e-6
  //  absolute_tolerance = 1e-6
  //  verbose            = true
  cusp::monitor<float> monitor(b, 5000, 1e-6, 1e-6, false);
  int restart = 50;
  
  // solve the linear system A x = b
  clock_gettime(CLOCK_MONOTONIC, &exec_start);
  cusp::krylov::gmres(A, x, b, restart, monitor, M);
  clock_gettime(CLOCK_MONOTONIC, &exec_end);

  // copy results back and write out
  cusp::array1d<float, cusp::host_memory> x_host(x);
  write_vector(argv[3], x_host);

  if(monitor.converged())
    {
      std::cout << "Solver converged to " << monitor.relative_tolerance() << " relative tolerance";
      std::cout << " with residual norm " << monitor.residual_norm();
      std::cout << " after " << monitor.iteration_count() << " iterations" << std::endl;
    } else {
      std::cout << "Solver reached iteration limit " << monitor.iteration_limit() << " before converging";
      std::cout << " to " << monitor.relative_tolerance() << " relative tolerance " << std::endl;
    }

  long int copy_time = elapsed_time_ms(copy_start, copy_end);
  long int execution_time = elapsed_time_ms(exec_start, exec_end);
  std::cout << "Copy time (ms): " << copy_time << std::endl;
  std::cout << "Execution time (ms): " << execution_time << std::endl;
  
  return 0;
}
