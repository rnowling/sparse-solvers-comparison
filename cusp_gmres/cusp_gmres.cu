#include "hip/hip_runtime.h"
#include <cusp/csr_matrix.h>
#include <cusp/monitor.h>
#include <cusp/krylov/gmres.h>
#include <cusp/gallery/poisson.h>

#include <fstream>
#include <stdio>

void load_vector(const char* flname, cusp::array1d<float, cusp::host_memory>& v)
{
  std::ifstream input_data;
  input_data.open(flname);

  int n_elements;
  input_data >> n_elements;

  for(int i = 0; i < n_elements; i++)
    {
      float entry;
      input_data >> n_elements;
      n_elements.push_back(entry);
    }

  input_data.close();
}

void write_vector(const char* flname, cusp::array1d<float, cusp::host_memory>& v)
{
  std::ofstream output_data;
  output_data.open(flname, std::ofstream::out | std::ofstream::trunc);

  output_data << v.size() << std::endl;
  for(int i = 0; i < v.size(); i++)
    {
      output_data << v[i] << std::endl;
    }
  
  output_data.close();
  
}


int main(int argc, char** argv)
{
  if(argc != 4)
    {
      std::cout << "Usage: " << argv[0] << " <matrix_flname> <input_vector_flname> <output_vector_flname>" << std::endl;
    }
  
  // create an empty sparse matrix structure (CSR format)
  cusp::csr_matrix<int, float, cusp::device_memory> A;
  // read matrix
  cusp::io::read_matrix_market_file(A, argv[1]);

  // create empty array
  cusp::array1d<float, cusp::host_memory> b_host(A.num_rows, 0);
  // read vector
  load_vector(argv[2], &b_host);

  cusp::array1d<float, cusp::device_memory> b(b_host);  
  cusp::array1d<float, cusp::device_memory> x(A.num_rows, 0);

  // set stopping criteria:
  //  iteration_limit    = 5000
  //  relative_tolerance = 1e-6
  //  absolute_tolerance = 0
  //  verbose            = true
  cusp::monitor<float> monitor(b, 5000, 1e-6, 0, true);
  int restart = 50;
  // set preconditioner (identity)
  cusp::identity_operator<float, cusp::device_memory> M(A.num_rows, A.num_rows);
  // solve the linear system A x = b
  cusp::krylov::gmres(A, x, b,restart, monitor, M);

  cusp::array1d<float, cusp::host_memory> x_host(x);
  write_vector(argv[3], &x_host);
  
  return 0;
}
