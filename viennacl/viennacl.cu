#include "hip/hip_runtime.h"
#include <fstream>
#include <iostream>
#include <sys/time.h>

#include <map>
#include <vector>

#ifndef NDEBUG
#define BOOST_UBLAS_NDEBUG
#endif

#include <boost/numeric/ublas/matrix_sparse.hpp>

#ifndef VIENNACL_WITH_CUDA
#define VIENNACL_WITH_CUDA
#endif

#define VIENNACL_WITH_UBLAS 1 

#include <viennacl/scalar.hpp>
#include <viennacl/vector.hpp>
#include <viennacl/compressed_matrix.hpp>
#include <viennacl/linalg/cg.hpp>
#include <viennacl/linalg/gmres.hpp>
#include <viennacl/linalg/row_scaling.hpp>

using namespace boost::numeric;

void load_vector(const char* flname, std::vector<float> &v)
{
  std::ifstream input_data;
  input_data.open(flname);

  int n_elements;
  input_data >> n_elements;

  v.resize(n_elements);

  for(int i = 0; i < n_elements; i++)
    {
      float entry;
      input_data >> entry;
      v[i] = entry;
    }

  input_data.close();
}

void load_matrix(const char* filename, ublas::compressed_matrix<float> &cpu_sparse_matrix,
                 unsigned int &n_rows, unsigned int &n_cols)
{
  unsigned n_nnzs;
  char buffer[255];
  FILE* matrix_file = fopen(filename, "r");
  
  // skip comments
  do {
    fgets(buffer, 255, matrix_file);
  } while(buffer[0] == '%');
  
  // read matrix dimensions
  if (sscanf(buffer, "%u %u %u", &n_rows, &n_cols, &n_nnzs) != 3)
    {
      printf("Failed to parse matrix entry\n");
      printf("Line: '%s'\n", buffer);
      fclose(matrix_file);
      exit(1);
    }
  
  printf("Reading a %d x %d matrix\n", n_rows, n_cols);

  cpu_sparse_matrix.resize(n_rows, n_cols, false);
    
  // read in data
  printf("Reading data\n");
  while(fgets(buffer, 255, matrix_file) != NULL && !feof(matrix_file))
    {
      unsigned int row, col;
      float value;
      
      if (sscanf(buffer, "%u %u %f", &row, &col, &value) != 3)
        {
          printf("Failed to parse matrix entry\n");
          printf("Line: '%s'\n", buffer);
          fclose(matrix_file);
          exit(1);
        }
      
      // convert from 1-indexing to 0-indexing
      cpu_sparse_matrix(row - 1, col - 1) = value;
    }
  
  fclose(matrix_file);
    
  printf("Read data\n");
}

void write_vector(const char* flname, std::vector<float> &v)
{
  std::ofstream output_data;
  output_data.open(flname, std::ofstream::out | std::ofstream::trunc);

  output_data << v.size() << std::endl;
  for(int i = 0; i < v.size(); i++)
    {
      output_data << v[i] << std::endl;
    }
  
  output_data.close();  
}

long int elapsed_time_ms(struct timespec &start, struct timespec &end)
{
  return (end.tv_sec * 1000 + end.tv_nsec / (1000 * 1000)) -
    (start.tv_sec * 1000 + start.tv_nsec / (1000 * 1000));
}


int main(int argc, char** argv)
{
  const char* PRECOND_NONE = "none";
  const char* PRECOND_DIAG = "diag";
  
  if(argc != 5)
    {
      std::cout << "Usage: " << argv[0] << " <preconditioner> <matrix_flname> <input_vector_flname> <output_vector_flname>" << std::endl;
      std::cout << std::endl;
      std::cout << "Preconditioner can be one of: " << PRECOND_NONE << " " << PRECOND_DIAG << std::endl;
      return 1;
    }

  if(strcmp(argv[1], PRECOND_NONE) != 0 and strcmp(argv[1], PRECOND_DIAG) != 0)
    {
      std::cout << "Preconditioner must be one of: " << PRECOND_NONE << " " << PRECOND_DIAG << std::endl;
      return 1;
    }
  else
    {
      std::cout << "Using preconditioner: " << argv[1] << std::endl;
    }

  unsigned int n_rows, n_cols;
  ublas::compressed_matrix<float> A_host;
  load_matrix(argv[2], A_host, n_rows, n_cols);
  viennacl::compressed_matrix<float> A_gpu(n_rows, n_cols);

  std::vector<float> b_host;
  load_vector(argv[3], b_host);
  viennacl::vector<float> b_gpu(b_host.size());

  std::cout << "Read dimensions: " << n_rows << " " << n_cols << std::endl;
  std::cout << "CPU Matrix dimensions: " << A_host.size1() << " " << A_host.size2() << std::endl;
  std::cout << "GPU Matrix dimensions: " << A_gpu.size1() << " " << A_gpu.size2() << std::endl;
  std::cout << "Vector length : " << b_host.size() << std::endl;      

  struct timespec copy_start;
  struct timespec copy_end;
  struct timespec exec_start;
  struct timespec exec_end;

  std::cout << "Copying data" << std::endl;
  hipDeviceSynchronize();
  clock_gettime(CLOCK_MONOTONIC, &copy_start);
  copy(b_host.begin(), b_host.end(), b_gpu.begin());
  copy(A_host, A_gpu);
  hipDeviceSynchronize();
  clock_gettime(CLOCK_MONOTONIC, &copy_end);
  
  // set stopping criteria:
  //  iteration_limit    = 5000
  //  relative_tolerance = 1e-6
  //  absolute_tolerance = 1e-6
  //  verbose            = true
  viennacl::linalg::gmres_tag my_tag(1e-6, 5000, 50);
  
  // solve the linear system A x = b
  std::cout << "Solve system" << std::endl;
  hipDeviceSynchronize();
  clock_gettime(CLOCK_MONOTONIC, &exec_start);
  viennacl::vector<float> x_gpu;
  // set preconditioner
  if (strcmp(argv[1], PRECOND_DIAG) == 0)
    {
      viennacl::linalg::row_scaling< viennacl::compressed_matrix<float> > vcl_row_scaling(A_gpu, viennacl::linalg::row_scaling_tag());
      x_gpu = viennacl::linalg::solve(A_gpu, b_gpu, my_tag, vcl_row_scaling);
    }
  else if(strcmp(argv[1], PRECOND_NONE) == 0)
    {
      x_gpu = viennacl::linalg::solve(A_gpu, b_gpu, my_tag);
    }
  hipDeviceSynchronize();
  clock_gettime(CLOCK_MONOTONIC, &exec_end);

  // copy results back and write out
  std::vector<float> x_host(n_cols);
  copy(x_gpu.begin(), x_gpu.end(), x_host.begin());
  write_vector(argv[4], x_host);

  std::cout << "Solver converged to " << my_tag.error() << " relative tolerance";
  std::cout << " after " << my_tag.iters() << " iterations" << std::endl;

  long int copy_time = elapsed_time_ms(copy_start, copy_end);
  long int execution_time = elapsed_time_ms(exec_start, exec_end);
  std::cout << "Copy time (ms): " << copy_time << std::endl;
  std::cout << "Execution time (ms): " << execution_time << std::endl;
  
  return 0;
}
