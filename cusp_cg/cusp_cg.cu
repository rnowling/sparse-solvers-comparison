#include "hip/hip_runtime.h"
#include <cusp/csr_matrix.h>
#include <cusp/monitor.h>
#include <cusp/krylov/cg.h>
#include <cusp/io/matrix_market.h>
#include <cusp/precond/diagonal.h>

#include <fstream>
#include <iostream>
#include <sys/time.h>

void load_vector(const char* flname, cusp::array1d<float, cusp::host_memory> &v)
{
  std::ifstream input_data;
  input_data.open(flname);

  int n_elements;
  input_data >> n_elements;

  v.resize(n_elements);

  for(int i = 0; i < n_elements; i++)
    {
      float entry;
      input_data >> entry;
      v[i] = entry;
    }

  input_data.close();
}

void write_vector(const char* flname, cusp::array1d<float, cusp::host_memory> &v)
{
  std::ofstream output_data;
  output_data.open(flname, std::ofstream::out | std::ofstream::trunc);

  output_data << v.size() << std::endl;
  for(int i = 0; i < v.size(); i++)
    {
      output_data << v[i] << std::endl;
    }
  
  output_data.close();  
}

long int elapsed_time_ms(struct timespec &start, struct timespec &end)
{
  return (end.tv_sec * 1000 + end.tv_nsec / (1000 * 1000)) -
    (start.tv_sec * 1000 + start.tv_nsec / (1000 * 1000));
}


int main(int argc, char** argv)
{
  const char* PRECOND_NONE = "none";
  const char* PRECOND_DIAG = "diag";
  
  if(argc != 5)
    {
      std::cout << "Usage: " << argv[0] << " <preconditioner> <matrix_flname> <input_vector_flname> <output_vector_flname>" << std::endl;
      std::cout << std::endl;
      std::cout << "Preconditioner can be one of: " << PRECOND_NONE << " " << PRECOND_DIAG << std::endl;
      return 1;
    }

    if(strcmp(argv[1], PRECOND_NONE) != 0 and strcmp(argv[1], PRECOND_DIAG) != 0)
    {
      std::cout << "Preconditioner must be one of: " << PRECOND_NONE << " " << PRECOND_DIAG << std::endl;
      return 1;
    }
  else
    {
      std::cout << "Using preconditioner: " << argv[1] << std::endl;
    }

  
  // create an empty sparse matrix structure (CSR format)
  cusp::csr_matrix<int, float, cusp::host_memory> A_host;
  // read matrix
  cusp::io::read_matrix_market_file(A_host, argv[2]);

  // create empty array
  cusp::array1d<float, cusp::host_memory> b_host(A_host.num_cols, 0);
  // read vector
  load_vector(argv[3], b_host);

  std::cout << "Matrix dimensions: " << A_host.num_rows << " " << A_host.num_cols << std::endl;
  std::cout << "Vector length : " << b_host.size() << std::endl;      

  struct timespec copy_start;
  struct timespec copy_end;
  struct timespec exec_start;
  struct timespec exec_end;

  hipDeviceSynchronize();
  clock_gettime(CLOCK_MONOTONIC, &copy_start);
  cusp::csr_matrix<int, float, cusp::device_memory> A(A_host);
  cusp::array1d<float, cusp::device_memory> b(b_host);  
  cusp::array1d<float, cusp::device_memory> x(A.num_rows, 0);
  hipDeviceSynchronize();
  clock_gettime(CLOCK_MONOTONIC, &copy_end);
  
  // set stopping criteria:
  //  iteration_limit    = 5000
  //  relative_tolerance = 1e-6
  //  absolute_tolerance = 1e-6
  //  verbose            = false
  cusp::monitor<float> monitor(b, 5000, 1e-6, 1e-6, false);
  
  // solve the linear system A x = b
  hipDeviceSynchronize();
  clock_gettime(CLOCK_MONOTONIC, &exec_start);
  if (strcmp(argv[1], PRECOND_DIAG) == 0)
    {
      cusp::precond::diagonal<float, cusp::device_memory> M(A);
      cusp::krylov::cg(A, x, b, monitor, M);
    }
    else if(strcmp(argv[1], PRECOND_NONE) == 0)
    {
      cusp::identity_operator<float, cusp::device_memory> M(A.num_rows, A.num_cols);
      cusp::krylov::cg(A, x, b, monitor, M);
    }
  hipDeviceSynchronize();
  clock_gettime(CLOCK_MONOTONIC, &exec_end);

  // copy results back and write out
  cusp::array1d<float, cusp::host_memory> x_host(x);
  write_vector(argv[4], x_host);

  if(monitor.converged())
    {
      std::cout << "Solver converged to " << monitor.relative_tolerance() << " relative tolerance";
      std::cout << " with residual norm " << monitor.residual_norm();
      std::cout << " after " << monitor.iteration_count() << " iterations" << std::endl;
    } else {
      std::cout << "Solver reached iteration limit " << monitor.iteration_limit() << " before converging";
      std::cout << " to " << monitor.relative_tolerance() << " relative tolerance " << std::endl;
    }

  long int copy_time = elapsed_time_ms(copy_start, copy_end);
  long int execution_time = elapsed_time_ms(exec_start, exec_end);
  std::cout << "Copy time (ms): " << copy_time << std::endl;
  std::cout << "Execution time (ms): " << execution_time << std::endl;
  
  return 0;
}
